#include "hip/hip_runtime.h"
#include "add.cuh"

#define BLOCKSIZE 1024

__global__ void add_dVector_kernel(double *a, double *b, double *c, int n) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if (id < n)
		c[id] = a[id] + b[id];
}

int add_dVectors(dVector a, dVector b, dVector c) {
	if (a.len != b.len || a.len != c.len) {
		fprintf(stderr, "Vector length mismatch\n");
		return -1;
	}

	double *device_a, *device_b, *device_c;
	int sizeInBytes = a.len * sizeof(double);
	hipError_t cudaStatus;


	cudaStatus = hipMalloc(&device_a, sizeInBytes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "A: hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc(&device_b, sizeInBytes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "B: hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc(&device_c, sizeInBytes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "C: hipMalloc failed!\n");
		goto Error;
	}

	hipMemcpy(device_a, a.data, sizeInBytes, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "A: hipMemcpy to device failed!\n");
		goto Error;
	}

	hipMemcpy(device_b, b.data, sizeInBytes, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "B: hipMemcpy to device failed!\n");
		goto Error;
	}


	int gridSize = (int)ceil((float)a.len / BLOCKSIZE);
	fprintf(stdout, "gridSize=%d, blockSize=%d\n", gridSize, BLOCKSIZE);
	add_dVector_kernel<<<gridSize, BLOCKSIZE>>>(device_a, device_b, device_c, a.len);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "add_dVector_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	hipMemcpy(c.data, device_c, sizeInBytes, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "C: hipMemcpy to host failed!\n");
		goto Error;
	}

Error:
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);

	return (cudaStatus == hipSuccess)? 0 : -1;
}

void test_add(int n) {
	dVector a, b, c;
	dvector_init(a, n);
	dvector_init(b, n);
	dvector_init(c, n);

	for (int i = 0; i < n; ++i) {
		a.data[i] = .75;
		b.data[i] = .25;
	}

	add_dVectors(a, b, c);

	// every element should be 1
	int errors = 0;
	for (int i = 0; i < n; ++i) {
//		fprintf(stdout, "c[%d]=%f\n", i, c.data[i]);
		if (abs(1 - c.data[i]) > 1e-14) {
			++errors;
		}
	}
	if (errors > 0) {
		fprintf(stdout, "Errors in %d of %d elements\n", errors, n);
	}
	else {
		fprintf(stdout, "No errors in %d elements\n", n);
	}

	dvector_free(a);
	dvector_free(b);
	dvector_free(c);

}